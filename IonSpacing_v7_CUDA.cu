#include "hip/hip_runtime.h"
/*
Title: Finding Eqilibrium Positions of Ions in a Linear Chain
Version: 7
Author: Renyi Chen
Description: This program solves equilibrium positions of ions in a linear
	chain by guessing the values then slightly adjust it until convergence

	Tested up to: 1500

CUDA Version: Cuda compilation tools, release 11.0, V11.0.221
*/

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>

#include <chrono>

#define N 1000	//Number of Ions
#define THREAD_PER_BLOCK 32

void guess_gen(double guess[]);
void converge_test(double& alpha, double u_guess[], double u_calc[],
	double u_guess_backup[], double& residual_s, bool& converge);

//Partially Calculation
__global__ void uj_calc_block(double* u, double* u_block) {
	__shared__ double partial_sum[THREAD_PER_BLOCK];

	for (int j = 0; j < N / 2; j++) {
		int index = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;
		if ((j != index) && (index < N)) {
			if (j > index) {
				partial_sum[threadIdx.x] = (1 / ((u[j] - u[index]) * (u[j] - u[index])));
			}
			else {
				partial_sum[threadIdx.x] = -(1 / ((u[j] - u[index]) * (u[j] - u[index])));
			}
		}
		else {
			partial_sum[threadIdx.x] = 0;
		}

		__syncthreads();
		__syncthreads();

		for (int s = 1; s < blockDim.x; s *= 2) {
			if ((threadIdx.x % (2 * s) == 0) && ((threadIdx.x + s) < blockDim.x)) {
				partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
			}
			__syncthreads();
		}

		if (threadIdx.x == 0) {
			u_block[j * (N / THREAD_PER_BLOCK + 1) + blockIdx.x] = partial_sum[0];
		}
		__syncthreads();
	}
}

//Obtain New Values from the Partial Calculation
__global__ void uj_calc(double* u_block, double* uj_calc) {
	int calcIndex = blockIdx.x * 32 + threadIdx.x;

	uj_calc[calcIndex] = 0;
	uj_calc[N - calcIndex - 1] = 0;
	if (calcIndex < N / 2) {
		for (int i = 0; i < (N / (THREAD_PER_BLOCK)+1); i++) {
			uj_calc[calcIndex] += u_block[calcIndex * (N / THREAD_PER_BLOCK + 1) + i];
		}

		uj_calc[N - calcIndex - 1] = -uj_calc[calcIndex];
	}
}

int main() {

	//====================================
	auto start = std::chrono::high_resolution_clock::now();
	//====================================

	//====================================
	//host variables
	double* u_guess_h;
	double* u_calc_h;

	//device variables
	double* u_guess_d;
	double* uj_block_result;
	double* u_calculated_d;
	//====================================
	//host memory allocation
	u_guess_h = (double*)malloc(sizeof(double) * N);
	u_calc_h = (double*)malloc(sizeof(double) * N);

	//device memory allocation
	hipMalloc((void**)&u_guess_d, sizeof(double) * N);
	hipMalloc((void**)&uj_block_result, sizeof(double) * N * (N / (THREAD_PER_BLOCK)+1) / 2);
	hipMalloc((void**)&u_calculated_d, sizeof(double) * N);
	//====================================

	//====================================
	//host variables
	double u_temp[N] = { 0 }, u_guess_backup[N] = { 0 };
	bool converge = false;
	double starting_alpha, alpha, residual_s;

	starting_alpha = 0.1;
	alpha = starting_alpha;
	residual_s = 0;
	//====================================

	//====================================
	//generate initial guesses
	guess_gen(u_guess_h);
	//====================================
	//int iteration = 0;
	std::cout << "N = " << N << std::endl;
	//====================================
	while (!(converge)) {
		//iteration++;
		//std::cout << "iteration = " << iteration << std::endl;
		//saving guess backup
		for (int i = 0; i < N; i++) {
			u_guess_backup[i] = u_temp[i];
			u_temp[i] = u_guess_h[i];
		}

		hipMemcpy(u_guess_d, u_guess_h, sizeof(double) * N, hipMemcpyHostToDevice);
		uj_calc_block << <(N / THREAD_PER_BLOCK + 1), THREAD_PER_BLOCK >> > (u_guess_d, uj_block_result);
		uj_calc << <N / 64 + 1, 32 >> > (uj_block_result, u_calculated_d);
		hipMemcpy(u_calc_h, u_calculated_d, sizeof(double) * N, hipMemcpyDeviceToHost);

		converge_test(alpha, u_guess_h, u_calc_h, u_guess_backup, residual_s, converge);

		//if guess value enters a loop where convergence can't be achieve
		//then the calculation is restarted with smaller starting alpha
		if ((alpha < 1e-12) && (residual_s > 100)) {
			guess_gen(u_guess_h);
			residual_s = 0;
			starting_alpha = starting_alpha * 0.95;
			alpha = starting_alpha;
		}
		else if (alpha < 1e-18) {
			guess_gen(u_guess_h);
			residual_s = 0;
			starting_alpha = starting_alpha * 0.95;
			alpha = starting_alpha;
		}

		/*
		std::cout << "u[0] = " << std::setprecision(16) << u_guess_h[0] << std::endl;
		std::cout << "alpha = " << alpha << std::endl;
		std::cout << "residual_s = " << residual_s << std::endl;
		std::cout << "-----------------------------------------" << std::endl;
		*/
	}
		//====================================
		for (int i = 0; i < N; i++) {
			std::cout << std::setprecision(16) << u_calc_h[i] << ',';
		}
	
		//====================================
		//free host memory
		free(u_guess_h);

		//free device memory
		hipFree(u_guess_d);
		hipFree(uj_block_result);
		hipFree(u_calculated_d);
		//====================================

		//====================================
		auto end = std::chrono::high_resolution_clock::now();
		auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

		double duration_sec;
		duration_sec = duration;
		duration_sec = duration_sec * 1e-6;
		std::cout << '\n' << duration_sec << std::endl;
		//====================================
	
	return 0;
}

/*=================================================================
Title: converge_test
	Description: check to see if convergence has occure, if not
		modify the guess value accordiang to calculated values
	return: none
=================================================================*/

void converge_test(double& alpha, double u_guess[], double u_calc[],
	double u_guess_backup[], double& residual_s, bool& converge) {

	//=====================================================
	double difference = 1e-8;	//Solution Resolution
	double residual_calc = 0;
	double guess_calc_diff;

	double outOfRange = (u_guess[0] - u_calc[0]) * alpha;	//Variable to check precision limit
	//=====================================================

	//=====================================================
	//calculate the sum of guessing value and calculated value
	for (int i = 0; i < N / 2; i++) {
		guess_calc_diff = std::abs(u_guess[i] - u_calc[i]);
		residual_calc += guess_calc_diff;
	}

	//check for difference between calculated value and guessing value
	for (int i = 0; i < N; i++) {
		double check = (std::abs(std::abs(u_guess[i]) - std::abs(u_calc[i])));
		if (check >= difference) {
			break;
		}
		else if (i == (N - 1)) {
			converge = true;
			return;
		}
	}

	//=====================================================

	//=====================================================
	//if calculated residual is larger than the residual from last iteration
	//alpha is decreased, and guessing value is restored
	if ((residual_s != 0) && (residual_calc > residual_s)) {
		alpha = alpha * 0.9;

		for (int i = 0; i < N; i++) {
			u_guess[i] = u_guess_backup[i];
		}

		return;
	}
	//=====================================================

	//=====================================================
	//reset alpha, when alpha is too small, 
	//which makes the (difference * alpha) too small
	if ((residual_calc == residual_s) && (u_guess[0] == (u_guess[0] - outOfRange))) {
		alpha = 0.1;
	}
	//=====================================================

	//=====================================================
	//if residual is decreased and program did not converge
	//then the guess = alpha * guess + (1-alpha) * calculated value
	for (int i = 0; i < (N / 2); i++) {
		guess_calc_diff = (u_guess[i] - u_calc[i]) * alpha;
		u_guess[i] = u_guess[i] - guess_calc_diff;
	}
	//copied for the other half
	for (int i = 0; i < (N / 2); i++) {
		u_guess[N - 1 - i] = -u_guess[i];
	}
	//=====================================================

	//=====================================================
	//new residual
	residual_s = residual_calc;
	//=====================================================

}

/*=================================================================
Title: guess_gen
	Description: This function generates the initial guess
=================================================================*/
void guess_gen(double guess[]) {
	for (int i = 0; i < N / 2; i++) {
		guess[i] = i - N / 2;
		guess[N - i - 1] = -guess[i];
	}
}


